// Copyright (c) 2023 The Decred developers.
//
// Decred BLAKE3 midstate-based CUDA kernel

// Written and optimized by Dave Collins Sep 2023.


#include <hip/hip_runtime.h>
#include <stdint.h>

#if defined(_WIN32)
#define DLLEXPORT __declspec(dllexport)
#else
#define DLLEXPORT
#endif /* _WIN32 */

#define MAX_OUTPUT_RESULTS 32

// Written and optimized by Dave Collins Sep 2023.
#if (__CUDACC_VER_MAJOR__ >= 10) && (__CUDA_ARCH__ > 300)
#define ROTR(v, n) __funnelshift_rc((v), (v), n)
#else
#define ROTR(v, n) ((v) >> n) | ((v) << (32 - n))
#endif


__global__
void search(
    uint32_t *output,
    // Midstate.
    const uint32_t cv0,
    const uint32_t cv1,
    const uint32_t cv2,
    const uint32_t cv3,
    const uint32_t cv4,
    const uint32_t cv5,
    const uint32_t cv6,
    const uint32_t cv7,

    // Final 52 bytes of data.
    const uint32_t m0,
    const uint32_t m1,
    const uint32_t m2,
    // const uint32_t m3 : nonce
    const uint32_t m4,
    const uint32_t m5,
    const uint32_t m6,
    const uint32_t m7,
    const uint32_t m8,
    const uint32_t m9,
    const uint32_t m10,
    const uint32_t m11,
    const uint32_t m12)
{
    // Nonce.
    const uint32_t m3 = blockDim.x * blockIdx.x + threadIdx.x;

    // BLAKE3 init vectors.
    const uint32_t iv0 = 0x6a09e667ul;
    const uint32_t iv1 = 0xbb67ae85ul;
    const uint32_t iv2 = 0x3c6ef372ul;
    const uint32_t iv3 = 0xa54ff53aul;
    // const uint32_t iv4 = 0x510e527ful;
    // const uint32_t iv5 = 0x9b05688cul;
    // const uint32_t iv6 = 0x1f83d9abul;
    // const uint32_t iv7 = 0x5be0cd19ul;

    // Internal compression func state.
    uint32_t v0, v1, v2, v3, v4, v5, v6, v7;
    uint32_t v8, v9, v10, v11, v12, v13, v14, v15;

    // Do the initialization and first round together.
    // Round 1.
    v0 = cv0 + cv4 + m0; v12 = ROTR(v0, 16); v8 = iv0 + v12; v4 = ROTR(cv4 ^ v8, 12); v0 += v4 + m1; v12 = ROTR(v12 ^ v0, 8); v8 += v12; v4 = ROTR(v4 ^ v8, 7);
    v1 = cv1 + cv5 + m2; v13 = ROTR(v1, 16); v9 = iv1 + v13; v5 = ROTR(cv5 ^ v9, 12); v1 += v5 + m3; v13 = ROTR(v13 ^ v1, 8); v9 += v13; v5 = ROTR(v5 ^ v9, 7);
    v2 = cv2 + cv6 + m4; v14 = ROTR(52 ^ v2, 16); v10 = iv2 + v14; v6 = ROTR(cv6 ^ v10, 12); v2 += v6 + m5; v14 = ROTR(v14 ^ v2, 8); v10 += v14; v6 = ROTR(v6 ^ v10, 7);
    v3 = cv3 + cv7 + m6; v15 = ROTR(10 ^ v3, 16); v11 = iv3 + v15; v7 = ROTR(cv7 ^ v11, 12); v3 += v7 + m7; v15 = ROTR(v15 ^ v3, 8); v11 += v15; v7 = ROTR(v7 ^ v11, 7);
    v0 = v0 + v5 + m8; v15 = ROTR(v15 ^ v0, 16); v10 += v15; v5 = ROTR(v5 ^ v10, 12); v0 += v5 + m9; v15 = ROTR(v15 ^ v0, 8); v10 += v15; v5 = ROTR(v5 ^ v10, 7);
    v1 = v1 + v6 + m10; v12 = ROTR(v12 ^ v1, 16); v11 += v12; v6 = ROTR(v6 ^ v11, 12); v1 += v6 + m11; v12 = ROTR(v12 ^ v1, 8); v11 += v12; v6 = ROTR(v6 ^ v11, 7);
    v2 = v2 + v7 + m12; v13 = ROTR(v13 ^ v2, 16); v8 += v13; v7 = ROTR(v7 ^ v8, 12); v2 += v7; v13 = ROTR(v13 ^ v2, 8); v8 += v13; v7 = ROTR(v7 ^ v8, 7);
    v3 = v3 + v4; v14 = ROTR(v14 ^ v3, 16); v9 += v14; v4 = ROTR(v4 ^ v9, 12); v3 += v4; v14 = ROTR(v14 ^ v3, 8); v9 += v14; v4 = ROTR(v4 ^ v9, 7);

    // Round 2 with message word permutation.
    v0 = v0 + v4 + m2; v12 = ROTR(v12 ^ v0, 16); v8 = v8 + v12; v4 = ROTR(v4 ^ v8, 12); v0 += v4 + m6; v12 = ROTR(v12 ^ v0, 8); v8 += v12; v4 = ROTR(v4 ^ v8, 7);
    v1 = v1 + v5 + m3; v13 = ROTR(v13 ^ v1, 16); v9 = v9 + v13; v5 = ROTR(v5 ^ v9, 12); v1 += v5 + m10; v13 = ROTR(v13 ^ v1, 8); v9 += v13; v5 = ROTR(v5 ^ v9, 7);
    v2 = v2 + v6 + m7; v14 = ROTR(v14 ^ v2, 16); v10 += v14; v6 = ROTR(v6 ^ v10, 12); v2 += v6 + m0; v14 = ROTR(v14 ^ v2, 8); v10 += v14; v6 = ROTR(v6 ^ v10, 7);
    v3 = v3 + v7 + m4; v15 = ROTR(v15 ^ v3, 16); v11 += v15; v7 = ROTR(v7 ^ v11, 12); v3 += v7; v15 = ROTR(v15 ^ v3, 8); v11 += v15; v7 = ROTR(v7 ^ v11, 7);
    v0 = v0 + v5 + m1; v15 = ROTR(v15 ^ v0, 16); v10 += v15; v5 = ROTR(v5 ^ v10, 12); v0 += v5 + m11; v15 = ROTR(v15 ^ v0, 8); v10 += v15; v5 = ROTR(v5 ^ v10, 7);
    v1 = v1 + v6 + m12; v12 = ROTR(v12 ^ v1, 16); v11 += v12; v6 = ROTR(v6 ^ v11, 12); v1 += v6 + m5; v12 = ROTR(v12 ^ v1, 8); v11 += v12; v6 = ROTR(v6 ^ v11, 7);
    v2 = v2 + v7 + m9; v13 = ROTR(v13 ^ v2, 16); v8 += v13; v7 = ROTR(v7 ^ v8, 12); v2 += v7; v13 = ROTR(v13 ^ v2, 8); v8 += v13; v7 = ROTR(v7 ^ v8, 7);
    v3 = v3 + v4; v14 = ROTR(v14 ^ v3, 16); v9 += v14; v4 = ROTR(v4 ^ v9, 12); v3 += v4 + m8; v14 = ROTR(v14 ^ v3, 8); v9 += v14; v4 = ROTR(v4 ^ v9, 7);

    // Round 3 with message word permutation.
    v0 = v0 + v4 + m3; v12 = ROTR(v12 ^ v0, 16); v8 = v8 + v12; v4 = ROTR(v4 ^ v8, 12); v0 += v4 + m4; v12 = ROTR(v12 ^ v0, 8); v8 += v12; v4 = ROTR(v4 ^ v8, 7);
    v1 = v1 + v5 + m10; v13 = ROTR(v13 ^ v1, 16); v9 = v9 + v13; v5 = ROTR(v5 ^ v9, 12); v1 += v5 + m12; v13 = ROTR(v13 ^ v1, 8); v9 += v13; v5 = ROTR(v5 ^ v9, 7);
    v2 = v2 + v6; v14 = ROTR(v14 ^ v2, 16); v10 += v14; v6 = ROTR(v6 ^ v10, 12); v2 += v6 + m2; v14 = ROTR(v14 ^ v2, 8); v10 += v14; v6 = ROTR(v6 ^ v10, 7);
    v3 = v3 + v7 + m7; v15 = ROTR(v15 ^ v3, 16); v11 += v15; v7 = ROTR(v7 ^ v11, 12); v3 += v7; v15 = ROTR(v15 ^ v3, 8); v11 += v15; v7 = ROTR(v7 ^ v11, 7);
    v0 = v0 + v5 + m6; v15 = ROTR(v15 ^ v0, 16); v10 += v15; v5 = ROTR(v5 ^ v10, 12); v0 += v5 + m5; v15 = ROTR(v15 ^ v0, 8); v10 += v15; v5 = ROTR(v5 ^ v10, 7);
    v1 = v1 + v6 + m9; v12 = ROTR(v12 ^ v1, 16); v11 += v12; v6 = ROTR(v6 ^ v11, 12); v1 += v6 + m0; v12 = ROTR(v12 ^ v1, 8); v11 += v12; v6 = ROTR(v6 ^ v11, 7);
    v2 = v2 + v7 + m11; v13 = ROTR(v13 ^ v2, 16); v8 += v13; v7 = ROTR(v7 ^ v8, 12); v2 += v7; v13 = ROTR(v13 ^ v2, 8); v8 += v13; v7 = ROTR(v7 ^ v8, 7);
    v3 = v3 + v4 + m8; v14 = ROTR(v14 ^ v3, 16); v9 += v14; v4 = ROTR(v4 ^ v9, 12); v3 += v4 + m1; v14 = ROTR(v14 ^ v3, 8); v9 += v14; v4 = ROTR(v4 ^ v9, 7);

    // Round 4 with message word permutation.
    v0 = v0 + v4 + m10; v12 = ROTR(v12 ^ v0, 16); v8 = v8 + v12; v4 = ROTR(v4 ^ v8, 12); v0 += v4 + m7; v12 = ROTR(v12 ^ v0, 8); v8 += v12; v4 = ROTR(v4 ^ v8, 7);
    v1 = v1 + v5 + m12; v13 = ROTR(v13 ^ v1, 16); v9 = v9 + v13; v5 = ROTR(v5 ^ v9, 12); v1 += v5 + m9; v13 = ROTR(v13 ^ v1, 8); v9 += v13; v5 = ROTR(v5 ^ v9, 7);
    v2 = v2 + v6; v14 = ROTR(v14 ^ v2, 16); v10 += v14; v6 = ROTR(v6 ^ v10, 12); v2 += v6 + m3; v14 = ROTR(v14 ^ v2, 8); v10 += v14; v6 = ROTR(v6 ^ v10, 7);
    v3 = v3 + v7; v15 = ROTR(v15 ^ v3, 16); v11 += v15; v7 = ROTR(v7 ^ v11, 12); v3 += v7; v15 = ROTR(v15 ^ v3, 8); v11 += v15; v7 = ROTR(v7 ^ v11, 7);
    v0 = v0 + v5 + m4; v15 = ROTR(v15 ^ v0, 16); v10 += v15; v5 = ROTR(v5 ^ v10, 12); v0 += v5 + m0; v15 = ROTR(v15 ^ v0, 8); v10 += v15; v5 = ROTR(v5 ^ v10, 7);
    v1 = v1 + v6 + m11; v12 = ROTR(v12 ^ v1, 16); v11 += v12; v6 = ROTR(v6 ^ v11, 12); v1 += v6 + m2; v12 = ROTR(v12 ^ v1, 8); v11 += v12; v6 = ROTR(v6 ^ v11, 7);
    v2 = v2 + v7 + m5; v13 = ROTR(v13 ^ v2, 16); v8 += v13; v7 = ROTR(v7 ^ v8, 12); v2 += v7 + m8; v13 = ROTR(v13 ^ v2, 8); v8 += v13; v7 = ROTR(v7 ^ v8, 7);
    v3 = v3 + v4 + m1; v14 = ROTR(v14 ^ v3, 16); v9 += v14; v4 = ROTR(v4 ^ v9, 12); v3 += v4 + m6; v14 = ROTR(v14 ^ v3, 8); v9 += v14; v4 = ROTR(v4 ^ v9, 7);

    // Round 5 with message word permutation.
    v0 = v0 + v4 + m12; v12 = ROTR(v12 ^ v0, 16); v8 = v8 + v12; v4 = ROTR(v4 ^ v8, 12); v0 += v4; v12 = ROTR(v12 ^ v0, 8); v8 += v12; v4 = ROTR(v4 ^ v8, 7);
    v1 = v1 + v5 + m9; v13 = ROTR(v13 ^ v1, 16); v9 = v9 + v13; v5 = ROTR(v5 ^ v9, 12); v1 += v5 + m11; v13 = ROTR(v13 ^ v1, 8); v9 += v13; v5 = ROTR(v5 ^ v9, 7);
    v2 = v2 + v6; v14 = ROTR(v14 ^ v2, 16); v10 += v14; v6 = ROTR(v6 ^ v10, 12); v2 += v6 + m10; v14 = ROTR(v14 ^ v2, 8); v10 += v14; v6 = ROTR(v6 ^ v10, 7);
    v3 = v3 + v7; v15 = ROTR(v15 ^ v3, 16); v11 += v15; v7 = ROTR(v7 ^ v11, 12); v3 += v7 + m8; v15 = ROTR(v15 ^ v3, 8); v11 += v15; v7 = ROTR(v7 ^ v11, 7);
    v0 = v0 + v5 + m7; v15 = ROTR(v15 ^ v0, 16); v10 += v15; v5 = ROTR(v5 ^ v10, 12); v0 += v5 + m2; v15 = ROTR(v15 ^ v0, 8); v10 += v15; v5 = ROTR(v5 ^ v10, 7);
    v1 = v1 + v6 + m5; v12 = ROTR(v12 ^ v1, 16); v11 += v12; v6 = ROTR(v6 ^ v11, 12); v1 += v6 + m3; v12 = ROTR(v12 ^ v1, 8); v11 += v12; v6 = ROTR(v6 ^ v11, 7);
    v2 = v2 + v7 + m0; v13 = ROTR(v13 ^ v2, 16); v8 += v13; v7 = ROTR(v7 ^ v8, 12); v2 += v7 + m1; v13 = ROTR(v13 ^ v2, 8); v8 += v13; v7 = ROTR(v7 ^ v8, 7);
    v3 = v3 + v4 + m6; v14 = ROTR(v14 ^ v3, 16); v9 += v14; v4 = ROTR(v4 ^ v9, 12); v3 += v4 + m4; v14 = ROTR(v14 ^ v3, 8); v9 += v14; v4 = ROTR(v4 ^ v9, 7);

    // Round 6 with message word permutation.
    v0 = v0 + v4 + m9; v12 = ROTR(v12 ^ v0, 16); v8 = v8 + v12; v4 = ROTR(v4 ^ v8, 12); v0 += v4; v12 = ROTR(v12 ^ v0, 8); v8 += v12; v4 = ROTR(v4 ^ v8, 7);
    v1 = v1 + v5 + m11; v13 = ROTR(v13 ^ v1, 16); v9 = v9 + v13; v5 = ROTR(v5 ^ v9, 12); v1 += v5 + m5; v13 = ROTR(v13 ^ v1, 8); v9 += v13; v5 = ROTR(v5 ^ v9, 7);
    v2 = v2 + v6 + m8; v14 = ROTR(v14 ^ v2, 16); v10 += v14; v6 = ROTR(v6 ^ v10, 12); v2 += v6 + m12; v14 = ROTR(v14 ^ v2, 8); v10 += v14; v6 = ROTR(v6 ^ v10, 7);
    v3 = v3 + v7; v15 = ROTR(v15 ^ v3, 16); v11 += v15; v7 = ROTR(v7 ^ v11, 12); v3 += v7 + m1; v15 = ROTR(v15 ^ v3, 8); v11 += v15; v7 = ROTR(v7 ^ v11, 7);
    v0 = v0 + v5; v15 = ROTR(v15 ^ v0, 16); v10 += v15; v5 = ROTR(v5 ^ v10, 12); v0 += v5 + m3; v15 = ROTR(v15 ^ v0, 8); v10 += v15; v5 = ROTR(v5 ^ v10, 7);
    v1 = v1 + v6 + m0; v12 = ROTR(v12 ^ v1, 16); v11 += v12; v6 = ROTR(v6 ^ v11, 12); v1 += v6 + m10; v12 = ROTR(v12 ^ v1, 8); v11 += v12; v6 = ROTR(v6 ^ v11, 7);
    v2 = v2 + v7 + m2; v13 = ROTR(v13 ^ v2, 16); v8 += v13; v7 = ROTR(v7 ^ v8, 12); v2 += v7 + m6; v13 = ROTR(v13 ^ v2, 8); v8 += v13; v7 = ROTR(v7 ^ v8, 7);
    v3 = v3 + v4 + m4; v14 = ROTR(v14 ^ v3, 16); v9 += v14; v4 = ROTR(v4 ^ v9, 12); v3 += v4 + m7; v14 = ROTR(v14 ^ v3, 8); v9 += v14; v4 = ROTR(v4 ^ v9, 7);

    // Round 7 with message word permutation.
    v0 = v0 + v4 + m11; v12 = ROTR(v12 ^ v0, 16); v8 = v8 + v12; v4 = ROTR(v4 ^ v8, 12); v0 += v4; v12 = ROTR(v12 ^ v0, 8); v8 += v12; v4 = ROTR(v4 ^ v8, 7);
    v1 = v1 + v5 + m5; v13 = ROTR(v13 ^ v1, 16); v9 = v9 + v13; v5 = ROTR(v5 ^ v9, 12); v1 += v5 + m0; v13 = ROTR(v13 ^ v1, 8); v9 += v13; v5 = ROTR(v5 ^ v9, 7);
    v2 = v2 + v6 + m1; v14 = ROTR(v14 ^ v2, 16); v10 += v14; v6 = ROTR(v6 ^ v10, 12); v2 += v6 + m9; v14 = ROTR(v14 ^ v2, 8); v10 += v14; v6 = ROTR(v6 ^ v10, 7);
    v3 = v3 + v7 + m8; v15 = ROTR(v15 ^ v3, 16); v11 += v15; v7 = ROTR(v7 ^ v11, 12); v3 += v7 + m6; v15 = ROTR(v15 ^ v3, 8); v11 += v15; v7 = ROTR(v7 ^ v11, 7);
    v0 = v0 + v5; v15 = ROTR(v15 ^ v0, 16); v10 += v15; v5 = ROTR(v5 ^ v10, 12); v0 += v5 + m10; v15 = ROTR(v15 ^ v0, 8); v10 += v15; v5 = ROTR(v5 ^ v10, 7);
    v1 = v1 + v6 + m2; v12 = ROTR(v12 ^ v1, 16); v11 += v12; v6 = ROTR(v6 ^ v11, 12); v1 += v6 + m12; v12 = ROTR(v12 ^ v1, 8); v11 += v12; v6 = ROTR(v6 ^ v11, 7);
    v2 = v2 + v7 + m3; v13 = ROTR(v13 ^ v2, 16); v8 += v13; v7 = ROTR(v7 ^ v8, 12); v2 += v7 + m4; v13 = ROTR(v13 ^ v2, 8); v8 += v13; v7 = ROTR(v7 ^ v8, 7);
    v3 = v3 + v4 + m7; v14 = ROTR(v14 ^ v3, 16); v9 += v14; v4 = ROTR(v4 ^ v9, 12); v3 += v4; v14 = ROTR(v14 ^ v3, 8); v9 += v14; v4 = ROTR(v4 ^ v9, 7);

    // Finally the truncated 256-bit output is defined as:
    //
    // h'0 = v0^v8
    // h'1 = v1^v9
    // h'2 = v2^v10
    // h'3 = v3^v11
    // h'4 = v4^v12
    // h'5 = v5^v13
    // h'6 = v6^v14
    // h'7 = v7^v15
    //
    // Just check if the last word (32-bits) is zeroed and return back to the
    // miner to notify it that a potential solution was found so it can check it
    // against the target difficulty.

    // Debug code to print result of hashing function.
    // if (!((v7 ^ v15) & 0xFFFF0000)) {
    //     printf("hash on gpu %x %x %x %x %x %x %x %x\n",
    //         v0 ^ v8, v1 ^ v9, v2 ^ v10, v3 ^ v11,
    //         v4 ^ v12, v5 ^ v13, v6 ^ v14, v7 ^ v15);
    //     printf("nonce for hash on gpu %x\n", m3);
    // }

    if (v7 ^ v15)
        return;

    // Update nonce.
    uint32_t pos = atomicInc(&output[0], 0xffffffff)+1;
    if (pos > MAX_OUTPUT_RESULTS) return; // Bounds check output buffer.
    output[pos] = m3;
}


extern "C" {
__host__ DLLEXPORT void
decred_blake3_hash(const uint32_t dimgrid, const uint32_t threads, uint32_t *cv, uint32_t *m, uint32_t *out)
{
	search<<<dimgrid, threads>>>(
		out,
		cv[0], cv[1], cv[2], cv[3], cv[4], cv[5], cv[6], cv[7],
		m[0], m[1], m[2],
		// m3,
		m[4], m[5], m[6], m[7], m[8], m[9], m[10], m[11], m[12]

	);
}
}
